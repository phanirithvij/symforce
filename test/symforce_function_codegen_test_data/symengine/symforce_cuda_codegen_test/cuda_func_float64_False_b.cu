
#include <hip/hip_runtime.h>
// -----------------------------------------------------------------------------
// This file was autogenerated by symforce from template:
//     function/FUNCTION.cu.jinja
// Do NOT modify by hand.
// -----------------------------------------------------------------------------

namespace sym {

__host__ __device__ void CudaFuncFloat64FalseB(
    const double a, const double1& b, const double* const __restrict__ c,
    const double* const __restrict__ d, const double* const __restrict__ e,
    const double* const __restrict__ f, const double* const __restrict__ g,
    double* const __restrict__ a_out, double* const __restrict__ b_out,
    double* const __restrict__ c_out, double* const __restrict__ d_out,
    double* const __restrict__ e_out, double* const __restrict__ f_out) {
  // Total ops: 36

  // Intermediate terms (1)
  const double _tmp0 = g[static_cast<size_t>(0)];

  // Output terms (6)
  if (a_out != nullptr) {
    *a_out = a;
  }

  if (b_out != nullptr) {
    b_out[0] = b.x;
  }

  if (c_out != nullptr) {
    c_out[0] = c[0];
    c_out[1] = c[1];
    c_out[2] = c[2];
  }

  if (d_out != nullptr) {
    d_out[0] = d[0];
    d_out[2] = d[2];
    d_out[1] = d[1];
    d_out[3] = d[3];
  }

  if (e_out != nullptr) {
    e_out[0] = e[0];
    e_out[1] = e[1];
    e_out[2] = e[2];
    e_out[3] = e[3];
    e_out[4] = e[4];
  }

  if (f_out != nullptr) {
    f_out[0] = _tmp0 + f[0];
    f_out[6] = _tmp0 + f[6];
    f_out[12] = _tmp0 + f[12];
    f_out[18] = _tmp0 + f[18];
    f_out[24] = _tmp0 + f[24];
    f_out[30] = _tmp0 + f[30];
    f_out[1] = _tmp0 + f[1];
    f_out[7] = _tmp0 + f[7];
    f_out[13] = _tmp0 + f[13];
    f_out[19] = _tmp0 + f[19];
    f_out[25] = _tmp0 + f[25];
    f_out[31] = _tmp0 + f[31];
    f_out[2] = _tmp0 + f[2];
    f_out[8] = _tmp0 + f[8];
    f_out[14] = _tmp0 + f[14];
    f_out[20] = _tmp0 + f[20];
    f_out[26] = _tmp0 + f[26];
    f_out[32] = _tmp0 + f[32];
    f_out[3] = _tmp0 + f[3];
    f_out[9] = _tmp0 + f[9];
    f_out[15] = _tmp0 + f[15];
    f_out[21] = _tmp0 + f[21];
    f_out[27] = _tmp0 + f[27];
    f_out[33] = _tmp0 + f[33];
    f_out[4] = _tmp0 + f[4];
    f_out[10] = _tmp0 + f[10];
    f_out[16] = _tmp0 + f[16];
    f_out[22] = _tmp0 + f[22];
    f_out[28] = _tmp0 + f[28];
    f_out[34] = _tmp0 + f[34];
    f_out[5] = _tmp0 + f[5];
    f_out[11] = _tmp0 + f[11];
    f_out[17] = _tmp0 + f[17];
    f_out[23] = _tmp0 + f[23];
    f_out[29] = _tmp0 + f[29];
    f_out[35] = _tmp0 + f[35];
  }
}

}  // namespace sym